#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void init_kernel ()
{
    printf("Block[%d, %d], thread[%d, %d], Dims[%d, %d]\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, blockDim.x, blockDim.y);
}

int main (int argc, char *argv[])
{

	float *a;
    int num_thread_blocks_x = 1, num_thread_blocks_y = 3;
    int num_threads_per_block_x = 2, num_threads_per_block_y = 2;

    printf("choosing gridDims: %d, %d\n", num_thread_blocks_x, num_thread_blocks_y);
    printf("choosing blockDims: %d, %d\n\n", num_threads_per_block_x, num_threads_per_block_y);


    dim3 dimBlock(num_threads_per_block_x, num_threads_per_block_y);
    dim3 dimGrid(num_thread_blocks_x, num_thread_blocks_y);

    init_kernel<<<dimGrid, dimBlock>>>();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

	return 0;
}
