#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define TILE 32
#define NUM_THREADS_PER_BLOCK 1024

__global__ void init_kernel (float *a, int n, int t, int span)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;
    for(int i = id * span; i < (id + 1) * span; i++)
    {
        a[i] = (float)i / (n*n);
    }
}

__global__ void kernel(float *a, float *b, float *c, int n)
{
	int c_row = blockIdx.x * blockDim.y + threadIdx.y;
	unsigned mask = 0xffffffff;
	float x = 0;
	__shared__ float atile[TILE][TILE];
	__shared__ float btile[TILE];

	for (int i=0; i< n / TILE; i++)
    {
		atile[threadIdx.y][threadIdx.x] = a[c_row * n + i * TILE + threadIdx.x];
		btile[threadIdx.x] = b[(i * TILE) + threadIdx.x];

		__syncthreads();
        if((threadIdx.x % warpSize) == 0)x += atile[threadIdx.y][threadIdx.x] * btile[threadIdx.x];
		else x = atile[threadIdx.y][threadIdx.x] * btile[threadIdx.x];

		__syncthreads();
		for (int j=warpSize/2; j>0; j=j/2) x += __shfl_down_sync(mask, x, j);
		__syncthreads();
	}

	if ((threadIdx.x % warpSize) == 0) c[c_row] = x;
	__syncthreads();
}

int main(int argc, char *argv[]){
    float *a, *b, *c;
	int n, t;
	struct timeval tv0, tv2;
	struct timezone tz0, tz2;
	n = atoi(argv[1]);
	t = atoi(argv[2]);

    if(t >= n * n)
    {
        t = n * n;
    }

    hipMallocManaged((void**)&a, sizeof(float)*n*n);
	hipMallocManaged((void**)&b, sizeof(float)*n);
	hipMallocManaged((void**)&c, sizeof(float)*n);

    int device;
    hipGetDevice(&device);

	hipMemAdvise(a, sizeof(float) * n * n, hipMemAdviseSetPreferredLocation, device);
	hipMemAdvise(b, sizeof(float) * n, hipMemAdviseSetPreferredLocation, device);
	hipMemAdvise(c, sizeof(float) * n, hipMemAdviseSetPreferredLocation, device);

    int span = (n * n) / t;
    if(t >= 1024)
    {
        init_kernel<<<t / 1024, 1024>>>(a, n, t, span);
    }
    else
    {
        init_kernel<<<1, t>>>(a, n, t, span);
    }
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
	hipDeviceSynchronize();
    int tc_n = t;
    if(t >= n)
    {
        tc_n = n;
    }
    span = n / tc_n;
    if(tc_n >= 1024)
    {
        init_kernel<<<tc_n / 1024, 1024>>>(b, n, t, span);
    }
    else
    {
        init_kernel<<<1, tc_n>>>(b, n, tc_n, span);
    }
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }

	hipDeviceSynchronize();
	gettimeofday(&tv0, &tz0);

    if(t < NUM_THREADS_PER_BLOCK)
    {
        int left = (TILE * TILE) / t;
        int adjusted_block = TILE / left;
        dim3 dimBlock(TILE, adjusted_block);

		if(t > (n / adjusted_block) * (TILE * adjusted_block))
        {
			dim3 dimGrid(n/ adjusted_block);
    		kernel<<<dimGrid, dimBlock>>>(a, b, c, n);
		}
        else
        {
			dim3 dimGrid(t / (TILE * adjusted_block));
			for(int i = 0; i < n; i += (t / (TILE * adjusted_block)) * adjusted_block)
            {
    			kernel<<<dimGrid, dimBlock>>>(a + i * n, b, c + i, n);
			}

        }
    }
    else
    {
        dim3 dimBlock(TILE, TILE);
		if(t > n * TILE)
        {
			dim3 dimGrid(n / TILE);
    		kernel<<<dimGrid, dimBlock>>>(a, b, c, n);
		}
		else
        {
			dim3 dimGrid(t / (TILE * TILE));
			for(int i = 0; i < n; i += (t/(TILE * TILE)) * TILE)
            {
    			kernel<<<dimGrid, dimBlock>>>(a + i * n, b, c + i, n);
			}
        }
    }
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
	gettimeofday(&tv2, &tz2);

	srand(time(0));
	int rowC = random() % n;
	float x = 0;
	for (int i=0; i<n; i++) x += a[rowC*n + i]*b[i];
	float error = fabs(c[rowC] - x);
	printf("Error: %0.12f, computed value: %0.12f, actual value: %0.12f, time: %ld microseconds\n", error, c[rowC], x, (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec));
    return 0;
}
