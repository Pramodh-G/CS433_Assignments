#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#define THREADS_PER_BLOCK 1024
#define TILE_SIZE 16

__global__ void init_kernel (float *a, int n)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    a[id] = (float)id / (n * n);
}

__global__ void kernel(float *a, float*b, float*c, int n)
{

}

int main (int argc, char *argv[])
{

	float *a, *b, *c;
    int n, t;
    struct timeval tv0, tv2;
    struct timezone tz0, tz2;

    n = atoi(argv[1]);

	hipMallocManaged((void**)&a, sizeof(float) * (n) * (n));
	hipMallocManaged((void**)&b, sizeof(float) * (n));
	hipMallocManaged((void**)&c, sizeof(float) * (n));

    int device = -1;
    hipGetDevice(&device);
    hipMemAdvise(a, sizeof(float) * (n) * (n), hipMemAdviseSetPreferredLocation, device);
    hipMemAdvise(b, sizeof(float) * (n), hipMemAdviseSetPreferredLocation, device);
    hipMemAdvise(c, sizeof(float) * (n), hipMemAdviseSetPreferredLocation, device);

    if((n) * (n) >= THREADS_PER_BLOCK)
    {
        init_kernel<<<(n * n)/ THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(a, n);
    }
    else
    {
        init_kernel<<<1, (n) * (n)>>>(a, n);
    }

    if((n) >= THREADS_PER_BLOCK)
    {
        init_kernel<<<(n)/ THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(b, n);
    }
    else
    {
        init_kernel<<<1, (n)>>>(b, n);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
	hipDeviceSynchronize();

    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid(n / TILE_SIZE, n / TILE_SIZE);

    gettimeofday(&tv0, &tz0);
    kernel<<<dimGrid, dimBlock>>>(a, b, c, n);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }
	hipDeviceSynchronize();
    gettimeofday(&tv2, &tz2);

    printf("time: %ld microseconds\n", (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec));
	return 0;
}
