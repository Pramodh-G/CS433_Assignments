#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
// #include <cuPrintf.h>

// #define ROWS_A  (1<<13)
// #define COLS_A  (1<<13)
#define TILE_SIZE 16
// int N = 64;
int numThr = 1;

__global__ void init_kernel (float *a, int N)
{
	int id = threadIdx.x + blockIdx.x*blockDim.x;

	a[id] = (float)id/(N*N);
	// printf("{%d}", id);
}

__global__ void matvecmult_kernel (float *a, float *b, float *c, int N)
{
	int c_row = blockIdx.y*blockDim.y + threadIdx.y;
	int i, j;
	float x = 0;

	for (i=0; i<N/TILE_SIZE; i++) {
		for (j=0; j<TILE_SIZE; j++) {
			x += (a[c_row * N + i*TILE_SIZE + j]*b[i*TILE_SIZE + j]);
			// d[0] = as[threadIdx.y][0]; d[1] = bs[0]; d[2] = as[threadIdx.y][1]; d[3] = bs[1];
		}
	}

	c[c_row] = x;
}


int main(int argc, char *argv[]){
    float *a, *b, *c;
	struct timeval tv0, tv2;
	struct timezone tz0, tz2;
	srand(time(0));
	int N;
    hipMallocManaged((void**)&N, sizeof(int));
	N = atoi(argv[1]);

    hipMallocManaged((void**)&a, sizeof(float)*N*N);
	hipMallocManaged((void**)&b, sizeof(float)*N*1);
	hipMallocManaged((void**)&c, sizeof(float)*N*1);

    int device = -1;
        hipGetDevice(&device);
	hipMemAdvise(c, sizeof(float)*N, hipMemAdviseSetPreferredLocation, device);

    init_kernel<<<N*N/1024, 1024>>>(a, N);
	init_kernel<<<N/1024, 1024>>>(b, N);
	hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Failed to launch kernel (error code: %s)!\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();
	gettimeofday(&tv0, &tz0);

    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid(N/TILE_SIZE, N/TILE_SIZE);

    matvecmult_kernel<<<dimGrid, dimBlock>>>(a, b, c, N);
    hipDeviceSynchronize();
	gettimeofday(&tv2, &tz2);

	int rowC = random() % N;

	float x = 0;

	for (int i=0; i<N; i++) x += a[rowC*N + i]*b[i];
	float error = fabs(c[rowC] - x);
	printf("Error: %0.12f, computed value: %0.12f, actual value: %0.12f, time: %ld microseconds\n", error, c[rowC], x, (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec));

    return 0;
}
